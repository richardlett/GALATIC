#include "hip/hip_runtime.h"
/*******************************************
#include "GALATIC/include/CSR.cuh"
#include "GALATIC/include/dCSR.cuh"
#include "GALATIC/include/SemiRingInterface.h"
#include "GALATIC/source/device/Multiply.cuh"

Your "includes" probably needs to look something like the above, rather than what's below. 
*******************************************/

#include "include/CSR.cuh"
#include "include/dCSR.cuh"
#include "include/SemiRingInterface.h"
#include "include/TestSpGEMM.cuh"

#include "source/device/Multiply.cuh"

struct foo {
    double a;

    double b;
    short c;
};

struct foo2 {
    short h;
    double a;
    double b;
    double c;

    double d;
    short k;
};

struct Arith_SR : SemiRing<foo, foo2, double>
{
  __host__ __device__ double multiply(const foo& a, const foo2& b) const { return a.b * b.d; }
  __host__ __device__ double add(const double& a, const double& b)   const   { return a + b; }
   __host__ __device__  static double AdditiveIdentity()                  { return     0; }
};


int main() 
{
    CSR<Arith_SR::leftInput_t> input_A_CPU;
    CSR<Arith_SR::rightInput_t> input_B_CPU;

    CSR<Arith_SR::output_t> result_mat_CPU;
    
    dCSR<Arith_SR::leftInput_t> input_A_GPU;
    dCSR<Arith_SR::rightInput_t> input_B_GPU;

    dCSR<Arith_SR::output_t> result_mat_GPU;

    input_A_CPU.alloc(2,2,4);
    input_A_CPU.row_offsets[0] = 0;
    input_A_CPU.row_offsets[1] = 2;
    input_A_CPU.row_offsets[2] = 4;

    input_A_CPU.col_ids[0] = 0;
    input_A_CPU.col_ids[1] = 1;

    input_A_CPU.col_ids[2] = 0;
    input_A_CPU.col_ids[3] = 1;


    input_B_CPU.alloc(2,2,4);
    input_B_CPU.row_offsets[0] = 0;
    input_B_CPU.row_offsets[1] = 2;
    input_B_CPU.row_offsets[2] = 4;

    input_B_CPU.col_ids[0] = 0;
    input_B_CPU.col_ids[1] = 1;

    input_B_CPU.col_ids[2] = 0;
    input_B_CPU.col_ids[3] = 1;


    /* ...
       ... load data into input_A_CPU
       ...*/
    
   
    for (int i = 0; i < 4; i++) {
        foo f;
        foo2 g;
        f.b = g.d = i+1;
        input_A_CPU.data[i] = f;
        input_B_CPU.data[i] = g;
    }
     // [ [ 1,  2],
     //   [ 3 4 ] ]
     hipDeviceSynchronize();

    
    // Transfer input matrices onto GPU
    convert(input_A_GPU, input_A_CPU);
    convert(input_B_GPU, input_B_CPU);

    // load data into semiring struct. For this one, we don't need to do anything
    Arith_SR semiring;
    
    
    // Setup execution options, we'll skip the details for now.
    
    const int Threads = 128;
    const int BlocksPerMP = 1;
    const int NNZPerThread = 2;
    const int InputElementsPerThreads = 2;
    const int RetainElementsPerThreads = 1;
    const int MaxChunksToMerge = 16;
    const int MaxChunksGeneralizedMerge = 256; // MAX: 865
    const int MergePathOptions = 8;
    
    
    GPUMatrixMatrixMultiplyTraits DefaultTraits(Threads, BlocksPerMP, NNZPerThread,
                                                 InputElementsPerThreads, RetainElementsPerThreads,
                                                 MaxChunksToMerge, MaxChunksGeneralizedMerge, MergePathOptions );
    
    const bool Debug_Mode = true;
    // DefaultTraits.preferLoadBalancing = true;
     ExecutionStats stats;
    // stats.measure_all = false;
    
    for (int i =0; i < 1; i++){
    // Actually perform the matrix multiplicaiton
        ACSpGEMM::Multiply<Arith_SR>(input_A_GPU, input_B_GPU, result_mat_GPU, DefaultTraits, stats, Debug_Mode, semiring);
         hipDeviceSynchronize();
    }

    TestSpGEMM(input_A_GPU, input_B_GPU, semiring, [=] (const Arith_SR::output_t &a, const Arith_SR::output_t &b) { return std::abs(a-b) < 0.01; }, DefaultTraits);

    convert(result_mat_CPU, result_mat_GPU);

    hipDeviceSynchronize();

    for (int i =0; i < 4; i++) {
        std::cout << "nnz: " << i <<   " val " <<  result_mat_CPU.data[i] << std::endl;
    }
    
}